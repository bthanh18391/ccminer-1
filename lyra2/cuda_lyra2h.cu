#include "hip/hip_runtime.h"
/**
 * Lyra2 (v1) cuda implementation based on djm34 work
 * tpruvot@github 2015, Nanashi 08/2016 (from 1.8-r2)
 * Lyra2h implentation for HPP coin based on all the previous
 * djm34 2017
 **/

#include <stdio.h>
#include <memory.h>

#define TPB52 8
#define TPB30 160
#define TPB20 160


//#include "cuda_lyra2h_sm2.cuh"
#include "cuda_lyra2h_sm5.cuh"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#endif
static uint32_t *h_GNonces[16]; // this need to get fixed as the rest of that routine
static uint32_t *d_GNonces[16];

__constant__ uint32_t pTarget[8];

__constant__ static uint2 blake2b_IV_sm2[8] = {
	{ 0xf3bcc908, 0x6a09e667 },
	{ 0x84caa73b, 0xbb67ae85 },
	{ 0xfe94f82b, 0x3c6ef372 },
	{ 0x5f1d36f1, 0xa54ff53a },
	{ 0xade682d1, 0x510e527f },
	{ 0x2b3e6c1f, 0x9b05688c },
	{ 0xfb41bd6b, 0x1f83d9ab },
	{ 0x137e2179, 0x5be0cd19 }
};

#define reduceDuplexRow(rowIn, rowInOut, rowOut) { \
	for (int i = 0; i < 16; i++) { \
		for (int j = 0; j < 12; j++) \
			state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut]; \
		round_lyra_sm2(state); \
		for (int j = 0; j < 12; j++) \
			Matrix[j + 12 * i][rowOut] ^= state[j]; \
		Matrix[0 + 12 * i][rowInOut] ^= state[11]; \
		Matrix[1 + 12 * i][rowInOut] ^= state[0]; \
		Matrix[2 + 12 * i][rowInOut] ^= state[1]; \
		Matrix[3 + 12 * i][rowInOut] ^= state[2]; \
		Matrix[4 + 12 * i][rowInOut] ^= state[3]; \
		Matrix[5 + 12 * i][rowInOut] ^= state[4]; \
		Matrix[6 + 12 * i][rowInOut] ^= state[5]; \
		Matrix[7 + 12 * i][rowInOut] ^= state[6]; \
		Matrix[8 + 12 * i][rowInOut] ^= state[7]; \
		Matrix[9 + 12 * i][rowInOut] ^= state[8]; \
		Matrix[10+ 12 * i][rowInOut] ^= state[9]; \
		Matrix[11+ 12 * i][rowInOut] ^= state[10]; \
	} \
  }

#define absorbblock(in)  { \
	state[0] ^= Matrix[0][in]; \
	state[1] ^= Matrix[1][in]; \
	state[2] ^= Matrix[2][in]; \
	state[3] ^= Matrix[3][in]; \
	state[4] ^= Matrix[4][in]; \
	state[5] ^= Matrix[5][in]; \
	state[6] ^= Matrix[6][in]; \
	state[7] ^= Matrix[7][in]; \
	state[8] ^= Matrix[8][in]; \
	state[9] ^= Matrix[9][in]; \
	state[10] ^= Matrix[10][in]; \
	state[11] ^= Matrix[11][in]; \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
	round_lyra_sm2(state); \
  }


__device__ __forceinline__
static void round_lyra_sm2(uint2 *s)
{
	Gfunc(s[0], s[4], s[8], s[12]);
	Gfunc(s[1], s[5], s[9], s[13]);
	Gfunc(s[2], s[6], s[10], s[14]);
	Gfunc(s[3], s[7], s[11], s[15]);
	Gfunc(s[0], s[5], s[10], s[15]);
	Gfunc(s[1], s[6], s[11], s[12]);
	Gfunc(s[2], s[7], s[8], s[13]);
	Gfunc(s[3], s[4], s[9], s[14]);
}

__device__ __forceinline__
void reduceDuplexRowSetuph(const int rowIn, const int rowInOut, const int rowOut, uint2 state[16], uint2 Matrix[192][16])
{
#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
	for (int i = 0; i < 16; i++)
	{
#pragma unroll
		for (int j = 0; j < 12; j++)
			state[j] ^= Matrix[12 * i + j][rowIn] + Matrix[12 * i + j][rowInOut];

		round_lyra_sm2(state);

#pragma unroll
		for (int j = 0; j < 12; j++)
			Matrix[j + 180 - 12 * i][rowOut] = Matrix[12 * i + j][rowIn] ^ state[j];

		Matrix[0 + 12 * i][rowInOut] ^= state[11];
		Matrix[1 + 12 * i][rowInOut] ^= state[0];
		Matrix[2 + 12 * i][rowInOut] ^= state[1];
		Matrix[3 + 12 * i][rowInOut] ^= state[2];
		Matrix[4 + 12 * i][rowInOut] ^= state[3];
		Matrix[5 + 12 * i][rowInOut] ^= state[4];
		Matrix[6 + 12 * i][rowInOut] ^= state[5];
		Matrix[7 + 12 * i][rowInOut] ^= state[6];
		Matrix[8 + 12 * i][rowInOut] ^= state[7];
		Matrix[9 + 12 * i][rowInOut] ^= state[8];
		Matrix[10 + 12 * i][rowInOut] ^= state[9];
		Matrix[11 + 12 * i][rowInOut] ^= state[10];
	}
}



__global__ __launch_bounds__(256, 1)
void lyra2h_gpu_hash_32_sm2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint2 Mask[8] = {
		{ 0x00000020, 0x00000000 },{ 0x00000020, 0x00000000 },
		{ 0x00000020, 0x00000000 },{ 0x00000010, 0x00000000 },
		{ 0x00000010, 0x00000000 },{ 0x00000010, 0x00000000 },
		{ 0x00000080, 0x00000000 },{ 0x00000000, 0x01000000 }
	};
	if (thread < threads)
	{
		uint2 state[16];

#pragma unroll
		for (int i = 0; i<4; i++) {
			LOHI(state[i].x, state[i].y, g_hash[threads*i + thread]);
		} //password

#pragma unroll
		for (int i = 0; i<4; i++) {
			state[i + 4] = state[i];
		} //salt

#pragma unroll
		for (int i = 0; i<8; i++) {
			state[i + 8] = blake2b_IV_sm2[i];
		}

		// blake2blyra x2
		//#pragma unroll 24
		for (int i = 0; i<12; i++) {
			round_lyra_sm2(state);
		}

		for (int i = 0; i<8; i++)
			state[i] ^= Mask[i];


		for (int i = 0; i<12; i++) {
			round_lyra_sm2(state);
		}


		uint2 Matrix[192][16]; // not cool

							 // reducedSqueezeRow0
#pragma unroll 16
		for (int i = 0; i < 16; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) {
				Matrix[j + 180 - 12 * i][0] = state[j];
			}
			round_lyra_sm2(state);
		}

		// reducedSqueezeRow1
#pragma unroll 16
		for (int i = 0; i < 16; i++)
		{
#pragma unroll 12
			for (int j = 0; j<12; j++) {
				state[j] ^= Matrix[j + 12 * i][0];
			}
			round_lyra_sm2(state);
#pragma unroll 12
			for (int j = 0; j<12; j++) {
				Matrix[j + 180 - 12 * i][1] = Matrix[j + 12 * i][0] ^ state[j];
			}
		}

		reduceDuplexRowSetuph(1, 0, 2, state, Matrix);
		reduceDuplexRowSetuph(2, 1, 3, state, Matrix);
		reduceDuplexRowSetuph(3, 0, 4, state, Matrix);
		reduceDuplexRowSetuph(4, 3, 5, state, Matrix);
		reduceDuplexRowSetuph(5, 2, 6, state, Matrix);
		reduceDuplexRowSetuph(6, 1, 7, state, Matrix);
		reduceDuplexRowSetuph(7, 0, 8, state, Matrix);
		reduceDuplexRowSetuph(8, 3, 9, state, Matrix);
		reduceDuplexRowSetuph(9, 6, 10, state, Matrix);
		reduceDuplexRowSetuph(10, 1, 11, state, Matrix);
		reduceDuplexRowSetuph(11, 4, 12, state, Matrix);
		reduceDuplexRowSetuph(12, 7, 13, state, Matrix);
		reduceDuplexRowSetuph(13, 2, 14, state, Matrix);
		reduceDuplexRowSetuph(14, 5, 15, state, Matrix);

		uint32_t rowa;
		uint32_t prev = 15;
		uint32_t iterator = 0;

		for (int tau=0;tau<8;tau++) {
			for (uint32_t i = 0; i<16; i++) {
				rowa = state[0].x & 15;
				reduceDuplexRow(prev, rowa, iterator);
				prev = iterator;
				iterator = (iterator + 7) & 15;
			}
			for (uint32_t i = 0; i<16; i++) {
				rowa = state[0].x & 15;
				reduceDuplexRow(prev, rowa, iterator);
				prev = iterator;
				iterator = (iterator - 1) & 15;
			}
		}



		absorbblock(rowa);
		uint32_t nonce = startNounce + thread;
		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}
	} //thread
}


__global__ __launch_bounds__(64, 1)
void lyra2h_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint2x4 Mask[2] = {
		0x00000020UL, 0x00000000UL, 0x00000020UL, 0x00000000UL,
		0x00000020UL, 0x00000000UL, 0x00000010UL, 0x00000000UL,
		0x00000010UL, 0x00000000UL, 0x00000010UL, 0x00000000UL,
		0x00000080UL, 0x00000000UL, 0x00000000UL, 0x01000000UL
	};
	const uint2x4 blake2b_IV[2] = {
	0xf3bcc908lu, 0x6a09e667lu,
	0x84caa73blu, 0xbb67ae85lu,
	0xfe94f82blu, 0x3c6ef372lu,
	0x5f1d36f1lu, 0xa54ff53alu,
	0xade682d1lu, 0x510e527flu,
	0x2b3e6c1flu, 0x9b05688clu,
	0xfb41bd6blu, 0x1f83d9ablu,
	0x137e2179lu, 0x5be0cd19lu
    };
	if (thread < threads)
	{
		uint2x4 state[4];

		state[0].x = state[1].x = __ldg(&g_hash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&g_hash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&g_hash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&g_hash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<12; i++)
			round_lyra(state); //because 12 is not enough

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];


		for (int i = 0; i<12; i++)
			round_lyra(state); //because 12 is not enough

		((uint2x4*)DMatrix)[threads * 0 + thread] = state[0];
		((uint2x4*)DMatrix)[threads * 1 + thread] = state[1];
		((uint2x4*)DMatrix)[threads * 2 + thread] = state[2];
		((uint2x4*)DMatrix)[threads * 3 + thread] = state[3];
	}
}

__global__
//__launch_bounds__(TPB52, 1)
void lyra2h_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = __ldg(&DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x]);
		state[1] = __ldg(&DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x]);
		state[2] = __ldg(&DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x]);
		state[3] = __ldg(&DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x]);

		reduceDuplexh2(state, thread, threads);
		reduceDuplexRowSetuph2(1, 0, 2, state, thread, threads);
		reduceDuplexRowSetuph2(2, 1, 3, state, thread, threads);
		reduceDuplexRowSetuph2(3, 0, 4, state, thread, threads);
		reduceDuplexRowSetuph2(4, 3, 5, state, thread, threads);
		reduceDuplexRowSetuph2(5, 2, 6, state, thread, threads);
		reduceDuplexRowSetuph2(6, 1, 7, state, thread, threads);
		reduceDuplexRowSetuph2(7, 0, 8, state, thread, threads);
		reduceDuplexRowSetuph2(8, 3, 9, state, thread, threads);
		reduceDuplexRowSetuph2(9, 6, 10, state, thread, threads);
		reduceDuplexRowSetuph2(10, 1, 11, state, thread, threads);
		reduceDuplexRowSetuph2(11, 4, 12, state, thread, threads);
		reduceDuplexRowSetuph2(12, 7, 13, state, thread, threads);
		reduceDuplexRowSetuph2(13, 2, 14, state, thread, threads);
		reduceDuplexRowSetuph2(14, 5, 15, state, thread, threads);

		uint32_t rowa; // = WarpShuffle(state[0].x, 0, 4) & 7;

		uint32_t prev = 15;
		uint32_t iterator = 0;


for (uint32_t tau=0;tau<7;tau++) {
		
		for (uint32_t i = 0; i<16; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 15;
			reduceDuplexRowth2(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 7) & 15;
		}

		for (uint32_t i = 0; i<16; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 15;
			reduceDuplexRowth2(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 15;
		}
}

		for (uint32_t i = 0; i<16; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 15;
			reduceDuplexRowth2(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 7) & 15;
		}

		for (uint32_t i = 0; i<15; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 15;
			reduceDuplexRowth2(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 15;
		}

//}
		rowa = WarpShuffle(state[0].x, 0, 4) & 15;
		reduceDuplexRowt_8_v2h(prev,iterator,rowa, state, thread, threads);


		DMatrix[(0 * threads + thread) * blockDim.x + threadIdx.x] = state[0];
		DMatrix[(1 * threads + thread) * blockDim.x + threadIdx.x] = state[1];
		DMatrix[(2 * threads + thread) * blockDim.x + threadIdx.x] = state[2];
		DMatrix[(3 * threads + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__ __launch_bounds__(64, 1)
void lyra2h_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash, uint32_t *resNonces)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&((uint2x4*)DMatrix)[threads * 0 + thread]);
		state[1] = __ldg4(&((uint2x4*)DMatrix)[threads * 1 + thread]);
		state[2] = __ldg4(&((uint2x4*)DMatrix)[threads * 2 + thread]);
		state[3] = __ldg4(&((uint2x4*)DMatrix)[threads * 3 + thread]);

		for (int i = 0; i < 12; i++)
			round_lyra(state);
		uint32_t nonce = startNounce + thread;
		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}
/*
		g_hash[thread + threads * 0] = state[0].x;
		g_hash[thread + threads * 1] = state[0].y;
		g_hash[thread + threads * 2] = state[0].z;
		g_hash[thread + threads * 3] = state[0].w;
*/
	} //thread
}


__global__ __launch_bounds__(64, 1)
void lyra2h_gpu_hash_32_1_sm5(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	const uint2x4 blake2b_IV[2] = {
		{ { 0xf3bcc908, 0x6a09e667 },{ 0x84caa73b, 0xbb67ae85 },{ 0xfe94f82b, 0x3c6ef372 },{ 0x5f1d36f1, 0xa54ff53a } },
		{ { 0xade682d1, 0x510e527f },{ 0x2b3e6c1f, 0x9b05688c },{ 0xfb41bd6b, 0x1f83d9ab },{ 0x137e2179, 0x5be0cd19 } }
	};
	const uint2x4 Mask[2] = {
		0x00000020UL, 0x00000000UL, 0x00000020UL, 0x00000000UL,
		0x00000020UL, 0x00000000UL, 0x00000010UL, 0x00000000UL,
		0x00000010UL, 0x00000000UL, 0x00000010UL, 0x00000000UL,
		0x00000080UL, 0x00000000UL, 0x00000000UL, 0x01000000UL
	};
	if (thread < threads)
	{
		uint2x4 state[4];

		((uint2*)state)[0] = __ldg(&g_hash[thread]);
		((uint2*)state)[1] = __ldg(&g_hash[thread + threads]);
		((uint2*)state)[2] = __ldg(&g_hash[thread + threads * 2]);
		((uint2*)state)[3] = __ldg(&g_hash[thread + threads * 3]);

		state[1] = state[0];
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i < 12; i++)
			round_lyra(state); //because 12 is not enough

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

		for (int i = 0; i < 12; i++)
			round_lyra(state); //because 12 is not enough


		((uint2x4*)DMatrix)[0 * threads + thread] = state[0];
		((uint2x4*)DMatrix)[1 * threads + thread] = state[1];
		((uint2x4*)DMatrix)[2 * threads + thread] = state[2];
		((uint2x4*)DMatrix)[3 * threads + thread] = state[3];
	}
}

__global__ __launch_bounds__(8, 1)
void lyra2h_gpu_hash_32_2_sm5(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.y * blockIdx.x + threadIdx.y);

	if (thread < threads)
	{
		uint2 state[4];

		state[0] = __ldg(&DMatrix[(0 * threads + thread)*blockDim.x + threadIdx.x]);
		state[1] = __ldg(&DMatrix[(1 * threads + thread)*blockDim.x + threadIdx.x]);
		state[2] = __ldg(&DMatrix[(2 * threads + thread)*blockDim.x + threadIdx.x]);
		state[3] = __ldg(&DMatrix[(3 * threads + thread)*blockDim.x + threadIdx.x]);

		reduceDuplexV5h(state, thread, threads);

		uint32_t rowa; // = WarpShuffle(state[0].x, 0, 4) & 7;
		uint32_t prev = 15;
		uint32_t iterator = 0;
		for (int tau = 0; tau<7; tau++) {
			for (uint32_t i = 0; i<16; i++) {
				rowa = WarpShuffle(state[0].x, 0, 4) & 15;
				reduceDuplexRowV50h(prev, rowa, iterator, state, thread, threads);
				prev = iterator;
				iterator = (iterator + 7) & 15;
			}
			for (uint32_t i = 0; i<16; i++) {
				rowa = WarpShuffle(state[0].x, 0, 4) & 15;
				reduceDuplexRowV50h(prev, rowa, iterator, state, thread, threads);
				prev = iterator;
				iterator = (iterator - 1) & 15;
			}
		}

		for (uint32_t i = 0; i<16; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 15;
			reduceDuplexRowV50h(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator + 7) & 15;
		}
		for (uint32_t i = 0; i<15; i++) {
			rowa = WarpShuffle(state[0].x, 0, 4) & 15;
			reduceDuplexRowV50h(prev, rowa, iterator, state, thread, threads);
			prev = iterator;
			iterator = (iterator - 1) & 15;
		}

		rowa = WarpShuffle(state[0].x, 0, 4) & 15;
		reduceDuplexRowV50_8_v2h(prev, iterator, rowa, state, thread, threads);



		DMatrix[(0 * threads + thread)*blockDim.x + threadIdx.x] = state[0];
		DMatrix[(1 * threads + thread)*blockDim.x + threadIdx.x] = state[1];
		DMatrix[(2 * threads + thread)*blockDim.x + threadIdx.x] = state[2];
		DMatrix[(3 * threads + thread)*blockDim.x + threadIdx.x] = state[3];
	}
}

__global__ __launch_bounds__(64, 1)
void lyra2h_gpu_hash_32_3_sm5(uint32_t threads, uint32_t startNounce, uint2 *g_hash, uint32_t *resNonces)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint2x4 state[4];

		state[0] = __ldg4(&((uint2x4*)DMatrix)[0 * threads + thread]);
		state[1] = __ldg4(&((uint2x4*)DMatrix)[1 * threads + thread]);
		state[2] = __ldg4(&((uint2x4*)DMatrix)[2 * threads + thread]);
		state[3] = __ldg4(&((uint2x4*)DMatrix)[3 * threads + thread]);

		for (int i = 0; i < 12; i++)
			round_lyra(state);


		uint32_t nonce = startNounce + thread;
		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}

	}
}




__global__	__launch_bounds__(48, 1)
void lyra2h_gpu_hash_32_v3(uint32_t threads, uint32_t startNounce, uint2 *outputHash, uint32_t *resNonces)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	ulonglong4 state[4];


	const ulonglong4 blake2b_IV[2] = {
		{ 0x6a09e667f3bcc908,
		0xbb67ae8584caa73b,
		0x3c6ef372fe94f82b,
		0xa54ff53a5f1d36f1 },
		{ 0x510e527fade682d1,
		0x9b05688c2b3e6c1f,
		0x1f83d9abfb41bd6b,
		0x5be0cd19137e2179 } };

	const ulonglong4 Mask[2] = {
		{ 0x20,
		0x20,
		0x20,
		0x08 },
		{ 0x08,
		0x08,
		0x80,
		0x0100000000000000 }
	};

#if __CUDA_ARCH__ == 350 || __CUDA_ARCH__ == 370
	if (thread < threads)
#endif
	{

		((uint2*)state)[0] = __ldg(&outputHash[thread]);
		((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		((uint2*)state)[2] = __ldg(&outputHash[thread + threads * 2]);
		((uint2*)state)[3] = __ldg(&outputHash[thread + threads * 3]);

		state[1] = state[0];

		state[2] = ((ulonglong4*)blake2b_IV)[0];
		state[3] = ((ulonglong4*)blake2b_IV)[1];

		for (int i = 0; i<12; i++)
			round_lyra_v35(state);  //because 12 is not enough

		state[0] ^= Mask[0];
		state[1] ^= Mask[1];

		for (int i = 0; i<12; i++)
			round_lyra_v35(state);  //because 12 is not enough


		uint32_t ps1 = (8 * memshift * 7 + 64 * memshift * thread);


		for (int i = 0; i < 8; i++)
		{
			uint32_t s1 = ps1 - 8 * memshift * i;
			for (int j = 0; j < 3; j++)
				((ulonglong4*)(DMatrix35))[j + s1] = (state)[j];

			round_lyra_v35(state);
		}


		reduceDuplexV3(state, thread);

		reduceDuplexRowSetupV3(1, 0, 2, state, thread);
		reduceDuplexRowSetupV3(2, 1, 3, state, thread);
		reduceDuplexRowSetupV3(3, 0, 4, state, thread);
		reduceDuplexRowSetupV3(4, 3, 5, state, thread);
		reduceDuplexRowSetupV3(5, 2, 6, state, thread);
		reduceDuplexRowSetupV3(6, 1, 7, state, thread);

		uint32_t rowa;
		uint32_t prev = 7;
		uint32_t iterator = 0;
		for (int j = 0; j<4; j++) {
			for (uint32_t i = 0; i<8; i++) {
				rowa = ((uint2*)state)[0].x & 7;
				reduceDuplexRowtV3(prev, rowa, iterator, state, thread);
				prev = iterator;
				iterator = (iterator + 3) & 7;
			}

			for (uint32_t i = 0; i<8; i++) {
				rowa = ((uint2*)state)[0].x & 7;
				reduceDuplexRowtV3(prev, rowa, iterator, state, thread);
				prev = iterator;
				iterator = (iterator - 1) & 7;
			}

		}

		uint32_t shift = (memshift * rowa + 64 * memshift * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&((ulonglong4*)(DMatrix35))[j + shift]);

		for (int i = 0; i < 12; i++)
			round_lyra_v35(state);

		uint32_t nonce = startNounce + thread;
		if (((uint64_t*)state)[3] <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}

	} //thread
}



__global__	__launch_bounds__(4 * 16, 1)
void lyra2h_gpu_hash_32_ws2(uint32_t threads, uint32_t startNounce, uint2 *outputHash, uint32_t *resNonces)
{
#if __CUDA_ARCH__ > 300 

	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	uint32_t thread2 = thread >> 2;

	vectype state;
	vectype temp[16];


	const uint28 blake2b_IV[2] = {
		{ 0xf3bcc908, 0x6a09e667,
		0x84caa73b, 0xbb67ae85,
		0xfe94f82b, 0x3c6ef372,
		0x5f1d36f1, 0xa54ff53a },
		{ 0xade682d1, 0x510e527f,
		0x2b3e6c1f, 0x9b05688c,
		0xfb41bd6b, 0x1f83d9ab,
		0x137e2179, 0x5be0cd19 } };

	const uint28 padding[2] = {
		{ 0x20, 0x0,
		0x20, 0x0,
		0x20, 0x0,
		0x10, 0x0 },
		{ 0x10, 0x0,
		0x10, 0x0,
		0x80, 0x0,
		0x00, 0x01000000 } };

	//		state.x = __ldg(&((u64type*)outputHash)[4 * thread2 + (threadIdx.x % 4)]);

	state.x = __ldg(&((u64type*)outputHash)[thread2 + (threadIdx.x % 4) * threads]);
	state.y = state.x;

	state.z = ((u64type*)blake2b_IV)[threadIdx.x % 4];
	state.w = ((u64type*)blake2b_IV)[4 + threadIdx.x % 4];


	for (int i = 0; i<12; i++)
		round_lyra_v35_ws(state);

	state.x ^= ((u64type*)padding)[threadIdx.x % 4];
	state.y ^= ((u64type*)padding)[4 + threadIdx.x % 4];


	for (int i = 0; i<12; i++)
		round_lyra_v35_ws(state);

	uint32_t ps1 = (memshift * (Ncol - 1) + Nrow * Ncol * memshift * thread2);

	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 - memshift * i;
		((vectype*)(DMatrix35))[(threadIdx.x % 4) + s1] = state;
		temp[(15 - i)] = state;
		round_lyra_v35_ws(state);
	}

	reduceDuplex_ws2(state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(1, 0, 2, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(2, 1, 3, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(3, 0, 4, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(4, 3, 5, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(5, 2, 6, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(6, 1, 7, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(7, 0, 8, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(8, 3, 9, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(9, 6, 10, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(10, 1, 11, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(11, 4, 12, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(12, 7, 13, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass1(13, 2, 14, state, thread2, temp);
	reduceDuplexRowSetup_ws2_pass2(14, 5, 15, state, thread2, temp);

	uint32_t rowa;
	uint32_t prev = 15;
	uint32_t iterator = 0;
	for (int j = 0; j<8; j++) {
		for (int i = 0; i < Ncol; i++)
		{
			rowa = shuffle2t(state.x, 0, 4) & 15;
			reduceDuplexRow_ws2(prev, rowa, iterator, state, thread2, temp);
			prev = iterator;
			iterator = (iterator + 7) & 15;
		}
		for (int i = 0; i < Ncol; i++)
		{
			rowa = shuffle2t(state.x, 0, 4) & 15;
			if (i == 15 && j == 7)
				reduceDuplexRow_ws2_v2(prev, rowa, iterator, state, thread2, temp);
			else
				reduceDuplexRow_ws2(prev, rowa, iterator, state, thread2, temp);

			prev = iterator;
			iterator = (iterator - 1) & 15;
		}
	}

	uint32_t shift = (memshift * Ncol * rowa + Nrow * Ncol * memshift * thread2);
	vectype tmp2 = temp[15];//__ldg4t(&((vectype*)(DMatrix35))[(threadIdx.x % 4) + shift]);
	state.x ^= tmp2.x;
	state.y ^= tmp2.y;
	state.z ^= tmp2.z;

	for (int i = 0; i < 12; i++)
		round_lyra_v35_ws(state);

	//	((u64type*)outputHash)[4 * thread2 + (threadIdx.x % 4)] = state.x;

	if ((threadIdx.x % 4) == 3) {
		uint32_t nonce = startNounce + thread2;
		if (devectorize(state.x) <= ((uint64_t*)pTarget)[3]) {
			atomicMin(&resNonces[1], resNonces[0]);
			atomicMin(&resNonces[0], nonce);
		}
	}
#endif
}





__host__
void lyra2h_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{

	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_GNonces[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 2 * sizeof(uint32_t));
}

__host__
void lyra2h_cpu_init_sm2(int thr_id, uint32_t threads)
{

	// just assign the device pointer allocated in main loop
	hipMalloc(&d_GNonces[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 2 * sizeof(uint32_t));
}
  

__host__
void lyra2h_cpu_init_sm35(int thr_id, uint32_t threads, uint64_t *d_matrix)
{

	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix35), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
	hipMalloc(&d_GNonces[thr_id], 2 * sizeof(uint32_t));
	hipHostMalloc(&h_GNonces[thr_id], 2 * sizeof(uint32_t));
}


__host__
uint32_t lyra2h_getSecNonce(int thr_id, int num)
{
	uint32_t results[2];
	memset(results, 0xFF, sizeof(results));
	hipMemcpy(results, d_GNonces[thr_id], sizeof(results), hipMemcpyDeviceToHost);
	if (results[1] == results[0])
		return UINT32_MAX;
	return results[num];
}

__host__
void lyra2h_setTarget(const void *pTargetIn)
{
	hipMemcpyToSymbol(HIP_SYMBOL(pTarget), pTargetIn, 32, 0, hipMemcpyHostToDevice);
}


__host__
uint32_t lyra2h_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, bool gtx750ti)
{
	uint32_t result = UINT32_MAX;
	hipMemset(d_GNonces[thr_id], 0xff, 2 * sizeof(uint32_t));
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB52;

	if (device_sm[dev_id] == 500) 
		tpb = TPB52;
	if (device_sm[dev_id] == 200) 
		tpb = TPB52;
	
	uint32_t tpb3 = 256;

	dim3 grid1((threads * 4 + tpb - 1) / tpb);
	dim3 block1(4, tpb >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	dim3 grid3((threads + tpb3 - 1) / tpb3);
	dim3 block3(tpb3);
	
	if (device_sm[dev_id] >= 500)
	{
		lyra2h_gpu_hash_32_1 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash);

		lyra2h_gpu_hash_32_2 <<< grid1, block1, 768 *sizeof(uint2) * tpb >>> (threads, startNounce, d_hash);

		lyra2h_gpu_hash_32_3 <<< grid2, block2 >>> (threads, startNounce, (uint2*)d_hash, d_GNonces[thr_id]);
	}
	else if (device_sm[dev_id] == 350 || device_sm[dev_id] == 370)
	{

		uint32_t tpb35 = 16;
		dim3 grid35_ws(threads / tpb35);
		dim3 block35_ws(4 * tpb35);

		lyra2h_gpu_hash_32_ws2 << <grid35_ws, block35_ws >> > (threads, startNounce, (uint2*)d_hash, d_GNonces[thr_id]);

	}
	else 
		lyra2h_gpu_hash_32_sm2 <<< grid3, block3 >>> (threads, startNounce, d_hash, d_GNonces[thr_id]);



	// get first found nonce
	hipMemcpy(h_GNonces[thr_id], d_GNonces[thr_id], 1 * sizeof(uint32_t), hipMemcpyDeviceToHost);
	result = *h_GNonces[thr_id];

	return result;

}
